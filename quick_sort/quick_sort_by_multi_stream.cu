#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <random>

#define INSERTION_SORT 32
#define MAX_DEPTH 16
template<typename T>
int partition(T* data, int low, int high){
    int mid = low + (high - low) / 2;
    T pivot = data[mid];
    std::swap(data[mid], data[low]);
    while (low < high)
    {
        while(data[high] >= pivot && low<high){
            high--;
        }
        data[low] = data[high];
        while(data[low] <= pivot && low<high){
            low++;
        }
        data[high] = data[low];
    }
    
    data[low] = pivot;
    return low;
}
template<typename T>
void quick_sort_cpu(T* data, int low, int high){
    
    if(low < high){
        int i = partition(data, low ,high);
        quick_sort_cpu(data, low, i-1);
        quick_sort_cpu(data, i+1, high);
    }
}

template<typename T>
__device__ void swap(T* a, T* b) {
    T temp = *a;
    *a = *b;
    *b = temp;
}

template<typename T>
__device__ int partition_gpu(T* data, int low, int high) {
    T pivot = data[low];
    int i = low;
    int j = high;

    while (true) {
        while (data[i] < pivot) {
            i++;
        }
        while (data[j] > pivot) {
            j--;
        }
        if (i >= j) {
            return j;
        }
        swap(&data[i], &data[j]);
        i++;
        j--;
    }
}
template<typename T>
__device__ void selection_sort(T *data, int low, int high){
    for(int i=low; i<high; ++i){
        T min = data[i];
        int min_index = i;
        for(int j=i+1; j<=high; ++j){

            if(data[j] < min){
                min_index = j;
                min = data[j];
            }
        }
        if(low != min_index) {
            data[min_index] = data[i];
            data[i] = min;
        };
    }
}

template<typename T>
__global__ void quick_sort_gpu(T* data, int low, int high, int depth) {
    if (low < high) {
        if(depth >= MAX_DEPTH || high - low <=INSERTION_SORT) {
            selection_sort(data, low, high);
            return;
        }
        int pi = partition_gpu(data, low, high);

        hipStream_t left_stream, right_stream;
        hipStreamCreateWithFlags(&left_stream, hipStreamNonBlocking);
        // Launch new kernels for the two partitions
        quick_sort_gpu<<<1, 1, 0 ,left_stream>>>(data, low, pi, depth+1);
        hipStreamDestroy(left_stream);
        hipStreamCreateWithFlags(&right_stream, hipStreamNonBlocking);
        quick_sort_gpu<<<1, 1, 0,right_stream>>>(data, pi + 1, high, depth+1);
        hipStreamDestroy(right_stream);

    }
}


int main() {
     const int size = 10000;
    int *output_data_cpu = new int[size];
    int *output_data_gpu = new int[size];
    int* device_data;

    // Initialize random data
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(1, 10000);
    for (int i = 0; i < size; ++i) {
        output_data_cpu[i] = distribution(generator);
    }
    
    // Copy data to device
    // hipMalloc((void**)&device_data, size * sizeof(int));
    hipMalloc(&device_data, size*sizeof(int));
    hipMemcpy(device_data, output_data_cpu, size * sizeof(int), hipMemcpyHostToDevice);

    // Sort on CPU
    quick_sort_cpu(output_data_cpu, 0, size);

    // Sort on GPU
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    //warmup
    // quick_sort_gpu<<<1, 1>>>(device_data, 0, size - 1, 0);
    // hipDeviceSynchronize();
    float millionseconds;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    quick_sort_gpu<<<1, 1>>>(device_data, 0, size, 0);
    hipDeviceSynchronize();
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&millionseconds,start,end);
    hipDeviceSynchronize();
    printf("Elapsed Time: %f\n", millionseconds);

    // Copy sorted data back from device
    hipMemcpy(output_data_gpu, device_data, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_data);
    // Check if CPU and GPU results are the same
    bool success = true;
    for (int i = 0; i < size; i++) {
        if (fabs((output_data_cpu[i] - output_data_gpu[i])) > 1e-5) {
            success = false;
            printf(" i =%d, output_data_cpu = %d, output_data_npu = %d\n", i, output_data_cpu[i], output_data_gpu[i]);
            break;
        }
    }

    if(success) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i=0;i<10;i++){
            printf("%d ",output_data_gpu[i]);
        }
        printf("\n");
        for(int i=0;i<10;i++){
            printf("%d ",output_data_cpu[i]);
        }
        printf("\n");
    }
    delete[] output_data_cpu;
    return 0;
}