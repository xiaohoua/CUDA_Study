#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include<vector>
#include<limits>

#include<hip/hip_runtime.h>
#include<hip/hip_fp16.h>
// softmax = exp(xi) / sigma(exp(xi))

#define warpSize 32


template<typename T>
void softmax_cpu(void* src, void* dst, int row, int col){

    for(int i=0; i<row; i++){
        T sum=0;

        T max = std::numeric_limits<T>::lowest();
        T* ptr_src = static_cast<T*>(src);
        T* ptr_dst = static_cast<T*>(dst);
        for(int j=0; j<col; j++){
            if(ptr_src[i * col + j] > max) max = ptr_src[i * col + j];
        }
        for(int j=0; j<col; j++){
            sum += exp(float(ptr_src[i * col + j] - max));
        }
        for(int j=0; j<col; j++){
            ptr_dst[i * col + j] = exp(float(ptr_src[i * col + j] - max))/(float)sum;
        }
    }
}

template<typename T>
struct ReduceMaxFunctor{
    __device__ T operator()(int offset, T val){
        return max(val, __shfl_xor_sync(0xffffffff, val, offset));
    }
};

template<typename T>
struct ReduceSumFunctor{
    __device__ T operator()(int offset, T val){
        return val += __shfl_xor_sync(0xffffffff, val, offset);
    }
};

template<typename Functor, typename T>
__device__  T ReduceOp(int tid, T val){

    for(int i=warpSize/2; i>0; i>>=1){
        val = Functor()(i, val);
    }
    return val;
}
template<typename T>
__global__ void softmax_gpu(void* src, void* dst, int row, int col){
    //让一个block处理一行
    int tid =  threadIdx.y * blockDim.x + threadIdx.x; 
    int blockid = blockIdx.y * gridDim.x + blockIdx.x;
    int gtid =  blockid * (blockDim.x * blockDim.y) + tid; 

    int total_step = gridDim.x * blockDim.x;
    int block_step = blockDim.x * blockDim.y;

    int row_id = blockid;

    T* ptr_src = static_cast<T*>(src);
    T* ptr_dst = static_cast<T*>(dst);
    T max_val = ptr_src[0];
    T sum_val = 0;
    __shared__ T shared_max[32];
    __shared__ T shared_sum[32];

    for(int i=row_id; i<row; i+=row){
        
        if(tid>=col) return;

        
        //先把一整行的最大值规约到一个blockDim.x里  

        for(int j = tid; j<col; j+=block_step){
            max_val = max(max_val, ptr_src[i * col + j]);
        }
        if(gtid == 0){
            printf("max_val = %d\n", max_val);
        } 
        max_val = ReduceOp<ReduceMaxFunctor<T>, T>(tid, max_val);

        if (tid % 32 == 0) {
            shared_max[tid / 32] = max_val;
        }
        __syncthreads();
        if(gtid == 0){
            printf("shared_max = %d\n", shared_max[0]);
        } 
        if (tid < 32) {
            max_val = shared_max[tid];
        }
        if(gtid == 0){
            printf("max_val = %d\n", max_val);
        } 
        __syncthreads();
        max_val = ReduceOp<ReduceMaxFunctor<T>, T>(tid, max_val);
        if(tid==0){
            shared_max[0] = max_val;
        }
        __syncthreads();
        max_val = shared_max[0];

        for(int j = tid; j<col; j+=block_step){
            sum_val += exp(ptr_src[i * col + j] - max_val);
        }
        sum_val = ReduceOp<ReduceSumFunctor<T>, T>(tid, sum_val);
        
        if (tid % 32 == 0) {
            shared_sum[tid / 32] = sum_val;
        }
        __syncthreads();

        if (tid < 32) {
            sum_val = shared_sum[tid];
        }
        __syncthreads();

        
        sum_val = ReduceOp<ReduceSumFunctor<T>, T>(tid, sum_val);
        if(tid==0){
            shared_sum[0] = sum_val;
        }
        __syncthreads();
        sum_val = shared_sum[0];

        for(int j = tid; j<col; j+=block_step){
            ptr_dst[i * col + j] = exp(ptr_src[i * col + j] - max_val) / sum_val;
        }

        
    }
}


int main() {
    const int rows = 1001;
    const int cols = 2050;

    std::vector<float> input_data(rows * cols, 1);
    std::vector<float> output_data_cpu(rows * cols, 0);
    std::vector<float> output_data_gpu(rows * cols, 0);

    softmax_cpu<float>(input_data.data(), output_data_cpu.data(), rows, cols);

    dim3 grids(256, (rows/256 + 31) /32 *32); //(256, 32)
    // dim3 blocks(1024, 1); //正确
    // dim3 blocks(1000, 1); //正确
    // dim3 blocks(1025, 1); //错误
    dim3 blocks(128, 8); //(128, 32)，错误//128*8=1024保证结果正确，128*9就不行

    void* d_input_data, *d_output_data;
    hipMalloc((void**)&d_input_data, rows*cols*sizeof(float));
    hipMalloc((void**)&d_output_data, rows*cols*sizeof(float));
    hipMemcpy(d_input_data, input_data.data(),rows*cols*sizeof(float),hipMemcpyHostToDevice);
    
    softmax_gpu<float><<<grids, blocks>>>(d_input_data, d_output_data, rows, cols);

    hipMemcpy(output_data_gpu.data(),d_output_data, rows*cols*sizeof(float),hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_input_data);
    hipFree(d_output_data);

    // Check if CPU and GPU results are the same
    bool success = true;
    for (int i = 0; i < rows * cols; i++) {
        if (fabs(output_data_cpu[i] - output_data_gpu[i]) > 1e-5) {
            success = false;
            printf(" i =%d, output_data_cpu = %f, output_data_npu = %f\n", i, output_data_cpu[i], output_data_gpu[i]);
            break;
        }
    }

    if(success) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i=0;i<10;i++){
            printf("%lf ",output_data_gpu[i]);
        }
        printf("\n");
        for(int i=0;i<10;i++){
            printf("%lf ",output_data_cpu[i]);
        }
        printf("\n");
    }
    return 0;
}
int main() {
    const int rows = 1000;
    const int cols = 1024;

    std::vector<float> input_data(rows * cols, 1);
    std::vector<float> output_data_cpu(rows * cols, 0);
    std::vector<float> output_data_gpu(rows * cols, 0);

    softmax_cpu<float>(input_data.data(), output_data_cpu.data(), rows, cols);

    // dim3 grids(256, (rows/256 + 31) /32 *32); //(256, 32)
    dim3 grids(10,100);

    dim3 blocks(128, 8); 

    void* d_input_data, *d_output_data;
    hipMalloc((void**)&d_input_data, rows*cols*sizeof(float));
    hipMalloc((void**)&d_output_data, rows*cols*sizeof(float));
    hipMemcpy(d_input_data, input_data.data(),rows*cols*sizeof(float),hipMemcpyHostToDevice);
    
    softmax_gpu<float><<<grids, blocks>>>(d_input_data, d_output_data, rows, cols);

    hipMemcpy(output_data_gpu.data(),d_output_data, rows*cols*sizeof(float),hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_input_data);
    hipFree(d_output_data);

    // Check if CPU and GPU results are the same
    bool success = true;
    for (int i = 0; i < rows * cols; i++) {
        if (fabs(float(output_data_cpu[i] - output_data_gpu[i])) > 1e-5) {
            success = false;
            printf(" i =%d, output_data_cpu = %f, output_data_npu = %f\n", i, output_data_cpu[i], output_data_gpu[i]);
            break;
        }
    }

    if(success) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i=0;i<10;i++){
            printf("%lf ",(float)output_data_gpu[i]);
        }
        printf("\n");
        for(int i=0;i<10;i++){
            printf("%lf ",output_data_cpu[i]);
        }
        printf("\n");
    }
    return 0;
}