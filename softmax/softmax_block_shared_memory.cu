#include <cmath>
#include <iostream>
#include<vector>
#include<limits>

#include<hip/hip_runtime.h>
#include<hip/hip_fp16.h>
// softmax = exp(xi) / sigma(exp(xi))

#define warpSize 32


template<typename T>
void softmax_cpu(void* src, void* dst, int row, int col){

    for(int i=0; i<row; i++){
        T sum=0;

        T max = std::numeric_limits<T>::lowest();
        T* ptr_src = static_cast<T*>(src);
        T* ptr_dst = static_cast<T*>(dst);
        for(int j=0; j<col; j++){
            if(ptr_src[i * col + j] > max) max = ptr_src[i * col + j];
        }
        for(int j=0; j<col; j++){
            sum += exp(float(ptr_src[i * col + j] - max));
        }
        for(int j=0; j<col; j++){
            ptr_dst[i * col + j] = exp(float(ptr_src[i * col + j] - max))/(float)sum;
        }
    }
}

template<typename T>
struct ReduceMaxFunctor{
    __device__ T operator()(int offset, T val){
        return max(val, __shfl_xor_sync(0xffffffff, val, offset));
    }
};

template<typename T>
struct ReduceSumFunctor{
    __device__ T operator()(int offset, T val){
        return val += __shfl_xor_sync(0xffffffff, val, offset);
    }
};

template<typename Functor, typename T>
__device__  T ReduceOp(int tid, T val){

    for(int i=warpSize/2; i>0; i>>=1){
        val = Functor()(i, val);
    }
    return val;
}
template<typename T>
__global__ void softmax_gpu(void* src, void* dst, int row, int col){
    //让一个block处理一行
    int tid =  threadIdx.y * blockDim.x + threadIdx.x; 
    int blockid = blockIdx.y * gridDim.x + blockIdx.x;
    int gtid =  blockid * (blockDim.x * blockDim.y) + tid; 

    int block_step = blockDim.x * blockDim.y;

    int row_id = blockid;

    T* ptr_src = static_cast<T*>(src);
    T* ptr_dst = static_cast<T*>(dst);
    T max_val = -1e37;
    T sum_val = 0;
    

    for(int i=row_id; i<row; i+=row){
        
        if(tid>=col) return;

        // 
        __shared__ T shared_max[1024];
        __shared__ T shared_sum[1024];

        shared_max[tid] = ptr_src[i * col + tid];
        __syncthreads();
        //先把一整行的最大值规约到一个blocksize里 
        for(int j = tid; j<col; j+=block_step){
            shared_max[tid] = max(shared_max[tid], ptr_src[i * col + j]);
        }
        __syncthreads();
        for(int j = block_step/2; j>0; j>>=1){
            if(tid<j){
                shared_max[tid] = max(shared_max[tid], shared_max[tid + j]);
            }
            
            __syncthreads();
        }
        shared_sum[tid] = 0;
        __syncthreads();
        // 先把一整行的sum规约到一个blocksize里 
        for(int j = tid; j<col; j+=block_step){
            shared_sum[tid] += exp(ptr_src[i * col + j] - shared_max[0]);
            
        }
        __syncthreads();
        for (int stride = block_step / 2; stride > 0; stride >>= 1) {
            if (tid < stride) {
                shared_sum[tid] += shared_sum[tid + stride];
            }
            __syncthreads();
        }

        for(int j = tid; j<col; j+=block_step){
            ptr_dst[i * col + j] = exp(ptr_src[i * col + j] - shared_max[0]) / shared_sum[0];
        }

    }
}

int main() {
    const int rows = 1000;
    const int cols = 1024;

    std::vector<float> input_data(rows * cols, 1);
    std::vector<float> output_data_cpu(rows * cols, 0);
    std::vector<float> output_data_gpu(rows * cols, 0);

    softmax_cpu<float>(input_data.data(), output_data_cpu.data(), rows, cols);

    // dim3 grids(256, (rows/256 + 31) /32 *32); //(256, 32)
    dim3 grids(10,100);

    dim3 blocks(128, 8); 

    void* d_input_data, *d_output_data;
    hipMalloc((void**)&d_input_data, rows*cols*sizeof(float));
    hipMalloc((void**)&d_output_data, rows*cols*sizeof(float));
    hipMemcpy(d_input_data, input_data.data(),rows*cols*sizeof(float),hipMemcpyHostToDevice);
    
    softmax_gpu<float><<<grids, blocks>>>(d_input_data, d_output_data, rows, cols);

    hipMemcpy(output_data_gpu.data(),d_output_data, rows*cols*sizeof(float),hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_input_data);
    hipFree(d_output_data);

    // Check if CPU and GPU results are the same
    bool success = true;
    for (int i = 0; i < rows * cols; i++) {
        if (fabs(float(output_data_cpu[i] - output_data_gpu[i])) > 1e-5) {
            success = false;
            printf(" i =%d, output_data_cpu = %f, output_data_npu = %f\n", i, output_data_cpu[i], output_data_gpu[i]);
            break;
        }
    }

    if(success) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i=0;i<10;i++){
            printf("%lf ",(float)output_data_gpu[i]);
        }
        printf("\n");
        for(int i=0;i<10;i++){
            printf("%lf ",output_data_cpu[i]);
        }
        printf("\n");
    }
    return 0;
}