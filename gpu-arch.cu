#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int count;

    hipGetDeviceCount(&count);
    for (int i = 0; i < count; ++i) {
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << "\n";
        std::cout << "Major revision number: " << prop.major << "\n";
        std::cout << "Minor revision number: " << prop.minor << "\n";
    }
    return 0;
}